#include "hip/hip_runtime.h"
/*
Demo: CUDA program to compute the squares of the first N natural numners
*/

#include <stdio.h>
typedef float data_t;  // makes it easy to change type later

__global__ void square(data_t *d_in, data_t *d_out); 	// kernel function
														// note the use of __global__
void printArray( data_t a[], int num_elements );


int main(){
	const int num_elements = 64; 
	int block_size = sizeof(d_in) * num_elements;

	// set-up sample data to beb procesed 
	data_t h_in[num_elements];
	for ( int idx=0; idx < num_elements; idx++ )
		h_in[idx] = idx;

	// define device data pointers
	data_t *d_in = NULL;
	data_t *d_out = NULL;

	// allocate device memory
	hipMalloc( (void **) &d_in, block_size );
	hipMalloc( (void **) &d_out, block_size );

	// transfer data to device memory
	hipMemcpy( d_in, h_in, block_size, hipMemcpyHostToDevice );

	// launch kernel
	square <<< 1, num_elements >>> (d_in, d_out);

	// retrieve data from device memory
	data_t h_out[num_elements];
	hipMemcpy( h_out, d_out, block_size, hipMemcpyDeviceToHost );

	// free device memory
	hipFree( d_in );
	hipFree( d_out );

	// display results
	printArray( h_out, num_elements );
}


// kernel definition
__global__ void square(data_t *d_in, data_t *d_out){
	int t_idx = threadIdx.x; // get thread id 
	d_out[t_idx] = d_in[t_idx] * d_in[t_idx];
}

void printArray( data_t a[], int num_elements ){
	for( int idx = 0; idx < num_elements; idx++ )
		printf("%5.1f%c", a[idx], ((idx % 4) != 3) ? '\t' : '\n');
	printf("\n");
}
